#include "hip/hip_runtime.h"
#include <string>
#include <chrono>
#include <iostream>
#include <fstream>
#include "CudaGeneralDeriv.h"
#include "ClassicDeriv.h"
#ifdef USE_COMPILED
#include "general.cu"
#include "general_flipped.cu"
#endif

namespace jit_test {

std::string GenerateGeneralCudaKernel(ClassicDeriv cd, bool flipped);

CudaGeneralDeriv::CudaGeneralDeriv(ClassicDeriv cd, bool flipped) :
  classicDeriv(cd), flipped(flipped),
  kernelJit(GenerateGeneralCudaKernel(cd, flipped).c_str(), flipped ? "solve_general_flipped" : "solve_general" )
{ };

std::chrono::duration<long, std::nano> CudaGeneralDeriv::Solve(double *rateConst, double *state, double *deriv, ClassicDeriv cd) {
  hipDeviceptr_t drateConst, dstate, dderiv, dnumReact, dnumProd, dreactId, dprodId;

  for (int i = 0; i < NUM_SPEC * NUM_CELLS; ++i) deriv[i] = 0.0;

  // Save predefined variable for CUDA kernel
  int numcell, numrxn, numspec, maxreact, maxprod;
  numcell  = NUM_CELLS;
  numrxn   = NUM_RXNS;
  numspec  = NUM_SPEC;
  maxreact = MAX_REACT;
  maxprod  = MAX_PROD;

  // Allocate GPU memory
  CUDA_SAFE_CALL( hipMalloc(&drateConst, NUM_RXNS * NUM_CELLS * sizeof(double)) );
  CUDA_SAFE_CALL( hipMalloc(&dstate, NUM_SPEC * NUM_CELLS * sizeof(double)) );
  CUDA_SAFE_CALL( hipMalloc(&dderiv, NUM_SPEC * NUM_CELLS * sizeof(double)) );
  CUDA_SAFE_CALL( hipMalloc(&dnumReact, NUM_RXNS * sizeof(int)) );
  CUDA_SAFE_CALL( hipMalloc(&dnumProd, NUM_RXNS * sizeof(int)) );
  CUDA_SAFE_CALL( hipMalloc(&dreactId, NUM_RXNS * MAX_REACT * sizeof(int)) );
  CUDA_SAFE_CALL( hipMalloc(&dprodId, NUM_RXNS * MAX_PROD * sizeof(int)) );

  // copy to GPU
  CUDA_SAFE_CALL( hipMemcpyHtoD(drateConst, rateConst, NUM_RXNS * NUM_CELLS * sizeof(double)) );
  CUDA_SAFE_CALL( hipMemcpyHtoD(dstate, state, NUM_SPEC * NUM_CELLS * sizeof(double)) );
  CUDA_SAFE_CALL( hipMemcpyHtoD(dnumReact, cd.numReact, NUM_RXNS * sizeof(int)) );
  CUDA_SAFE_CALL( hipMemcpyHtoD(dnumProd, cd.numProd, NUM_RXNS * sizeof(int)) );
  CUDA_SAFE_CALL( hipMemcpyHtoD(dreactId, cd.reactId, NUM_RXNS * MAX_REACT * sizeof(int)) );
  CUDA_SAFE_CALL( hipMemcpyHtoD(dprodId, cd.prodId, NUM_RXNS * MAX_PROD * sizeof(int)) );

  // Call the function
  void *args[] = { &drateConst, &dstate, &dderiv, &dnumReact, &dnumProd, &dreactId,
                   &dprodId, &numcell, &numrxn, &numspec, &maxreact, &maxprod };

  auto start = std::chrono::high_resolution_clock::now();
  kernelJit.Run(args);
  auto stop = std::chrono::high_resolution_clock::now();

  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);

  // Get the result
  CUDA_SAFE_CALL( hipMemcpyDtoH(deriv, dderiv, NUM_SPEC * NUM_CELLS * sizeof(double)) );

  CUDA_SAFE_CALL( hipFree(drateConst) );
  CUDA_SAFE_CALL( hipFree(dstate) );
  CUDA_SAFE_CALL( hipFree(dderiv) );
  CUDA_SAFE_CALL( hipFree(dnumReact) );
  CUDA_SAFE_CALL( hipFree(dnumProd) );
  CUDA_SAFE_CALL( hipFree(dreactId) );
  CUDA_SAFE_CALL( hipFree(dprodId) );

  return time;
}

std::chrono::duration<long, std::nano> CudaGeneralDeriv::SolveCompiled(double *rateConst, double *state, double *deriv, ClassicDeriv cd) {
  double *drateConst, *dstate, *dderiv;
  int *dnumReact, *dnumProd, *dreactId, *dprodId;

  for (int i = 0; i < NUM_SPEC * NUM_CELLS; ++i) deriv[i] = 0.0;

  // Save predefined variable for CUDA kernel
  int numcell, numrxn, numspec, maxreact, maxprod;
  numcell  = NUM_CELLS;
  numrxn   = NUM_RXNS;
  numspec  = NUM_SPEC;
  maxreact = MAX_REACT;
  maxprod  = MAX_PROD;

  // Allocate GPU memory
  hipMalloc(&drateConst, NUM_RXNS * NUM_CELLS * sizeof(double));
  hipMalloc(&dstate, NUM_SPEC * NUM_CELLS * sizeof(double));
  hipMalloc(&dderiv, NUM_SPEC * NUM_CELLS * sizeof(double));
  hipMalloc(&dnumReact, NUM_RXNS * sizeof(int));
  hipMalloc(&dnumProd, NUM_RXNS * sizeof(int));
  hipMalloc(&dreactId, NUM_RXNS * MAX_REACT * sizeof(int));
  hipMalloc(&dprodId, NUM_RXNS * MAX_PROD * sizeof(int));

  // copy to GPU
  hipMemcpy(drateConst, rateConst, NUM_RXNS * NUM_CELLS * sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy(dstate, state, NUM_SPEC * NUM_CELLS * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dnumReact, cd.numReact, NUM_RXNS * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(dnumProd, cd.numProd, NUM_RXNS * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(dreactId, cd.reactId, NUM_RXNS * MAX_REACT * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(dprodId, cd.prodId, NUM_RXNS * MAX_PROD * sizeof(int), hipMemcpyHostToDevice );

  auto start = std::chrono::high_resolution_clock::now();
  // Call the function
#ifdef USE_COMPILED
  if (this->flipped) {
    solve_general_flipped<<<CUDA_BLOCKS,CUDA_THREADS>>>(drateConst, dstate, dderiv,
        dnumReact, dnumProd, dreactId, dprodId, numcell, numrxn, numspec, maxreact, maxprod);
  } else {
    solve_general<<<CUDA_BLOCKS,CUDA_THREADS>>>(drateConst, dstate, dderiv,
        dnumReact, dnumProd, dreactId, dprodId, numcell, numrxn, numspec, maxreact, maxprod);
  }
#endif
  auto stop = std::chrono::high_resolution_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);

  // Get the result
  hipMemcpy(deriv, dderiv, NUM_SPEC * NUM_CELLS * sizeof(double), hipMemcpyDeviceToHost );

  hipFree(drateConst);
  hipFree(dstate);
  hipFree(dderiv);
  hipFree(dnumReact);
  hipFree(dnumProd);
  hipFree(dreactId);
  hipFree(dprodId);

  return time;
}

void CudaGeneralDeriv::OutputCuda(const char *fileName) {
  std::ofstream outFile;
  outFile.open(fileName);
  outFile << GenerateGeneralCudaKernel(this->classicDeriv, this->flipped);
  outFile.close();
}

std::string GenerateGeneralCudaKernel(ClassicDeriv cd, bool flipped) {

  std::string kernel;
  if (!flipped) {
    kernel = "\n\
extern \"C\" __global__                                                     \n\
void solve_general(double *rateConst, double *state, double *deriv,                 \n\
           int *numReact, int *numProd, int *reactId, int *prodId,          \n\
           int numcell, int numrxn, int numspec, int maxreact, int maxprod) \n\
                                                                            \n\
{                                                                           \n\
  size_t tid;                                                               \n\
  int i_spec, i_rxn, i_react, i_prod;                                       \n\
  double rate;                                                              \n\
                                                                            \n\
  tid = blockIdx.x * blockDim.x + threadIdx.x;                              \n\
  if (tid < numcell) {                                                      \n\
     for (i_spec = 0; i_spec < numspec; ++i_spec)                           \n\
         deriv[i_spec+numspec*tid] = 0.0;                                   \n\
     for (i_rxn = 0; i_rxn < numrxn; ++i_rxn) {                             \n\
         rate = rateConst[i_rxn+numrxn*tid];                                \n\
         for (i_react = 0; i_react < numReact[i_rxn]; ++i_react)            \n\
             rate *= state[reactId[i_rxn*maxreact+i_react]+numspec*tid];    \n\
         for (i_react = 0; i_react < numReact[i_rxn]; ++i_react)            \n\
             deriv[reactId[i_rxn*maxreact+i_react]+numspec*tid] -= rate;    \n\
         for (i_prod = 0; i_prod < numProd[i_rxn]; ++i_prod)                \n\
             deriv[prodId[i_rxn*maxprod+i_prod]+numspec*tid] += rate;       \n\
     }                                                                      \n\
  }                                                                         \n\
}                                                                           \n";
  } else {
    kernel = "\n\
extern \"C\" __global__                                                     \n\
void solve_general_flipped(double *rateConst, double *state, double *deriv,                 \n\
           int *numReact, int *numProd, int *reactId, int *prodId,          \n\
           int numcell, int numrxn, int numspec, int maxreact, int maxprod) \n\
                                                                            \n\
{                                                                           \n\
  size_t tid;                                                               \n\
  int i_spec, i_rxn, i_react, i_prod;                                       \n\
  double rate;                                                              \n\
                                                                            \n\
  tid = blockIdx.x * blockDim.x + threadIdx.x;                              \n\
  if (tid < numcell) {                                                      \n\
     for (i_spec = 0; i_spec < numspec; ++i_spec)                           \n\
         deriv[i_spec*numcell+tid] = 0.0;                                   \n\
     for (i_rxn = 0; i_rxn < numrxn; ++i_rxn) {                             \n\
         rate = rateConst[i_rxn*numcell+tid];                               \n\
         for (i_react = 0; i_react < numReact[i_rxn]; ++i_react)            \n\
             rate *= state[reactId[i_rxn*maxreact+i_react]*numcell+tid];    \n\
         for (i_react = 0; i_react < numReact[i_rxn]; ++i_react)            \n\
             deriv[reactId[i_rxn*maxreact+i_react]*numcell+tid] -= rate;    \n\
         for (i_prod = 0; i_prod < numProd[i_rxn]; ++i_prod)                \n\
             deriv[prodId[i_rxn*maxprod+i_prod]*numcell+tid] += rate;       \n\
     }                                                                      \n\
  }                                                                         \n\
}                                                                           \n";
  }
  return kernel;
}


} // namespace jit_test

